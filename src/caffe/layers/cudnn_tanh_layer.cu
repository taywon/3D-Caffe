
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNTanHLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  cudnnActivationDescriptor_t *activationDesc;
  CUDNN_CHECK(cudnnCreateActivationDescriptor(activationDesc ));
  CUDNN_CHECK(cudnnSetActivationDescriptor(*activationDesc,CUDNN_ACTIVATION_TANH,CUDNN_NOT_PROPAGATE_NAN,0));
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        *activationDesc,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNTanHLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    cudnnActivationDescriptor_t *activationDesc;
  CUDNN_CHECK(cudnnCreateActivationDescriptor(activationDesc ));
  CUDNN_CHECK(cudnnSetActivationDescriptor(*activationDesc,CUDNN_ACTIVATION_TANH,CUDNN_NOT_PROPAGATE_NAN,0));

  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        *activationDesc,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNTanHLayer);

}  // namespace caffe
#endif
